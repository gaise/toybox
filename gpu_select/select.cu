#include <iostream>
#include <hip/hip_runtime.h>
#include <random>

#define N 4096
#define THREAD 256
#define BLOCK 18

#define HANDLE_ERROR(x) checkCudaError(x, __LINE__)

void checkCudaError(hipError_t msg, int x)
{
  if (msg != hipSuccess) {
    fprintf(stderr, "line: %d %s\n", x, hipGetErrorString(msg));
    exit(1);
  }
  return;
}

__global__ void kernel(double *x, double *y, double *z, double *tf, double *ox, double *oy, double *oz)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = id; i < N; i+=stride)
		{
			ox[i] = x[i] * tf[0] + y[i] * tf[1] + z[i] * tf[2] + tf[3];
			oy[i] = x[i] * tf[4] + y[i] * tf[5] + z[i] * tf[6] + tf[7];
			oz[i] = x[i] * tf[8] + y[i] * tf[9] + z[i] * tf[10] + tf[11];
		}
}

int main()
{
	hipEvent_t start[2], stop[2];
	float time;

	double org_x[N], org_y[N], org_z[N];

	std::random_device rnd;
	std::mt19937 mt(rnd());
	std::uniform_real_distribution<double> rand100(0.0, 100.0);

	for (int i = 0; i < N; i++)
		{
			org_x[i] = rand100(mt);
			org_y[i] = rand100(mt);
			org_z[i] = rand100(mt);
		}
	
	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));

	double tf_org[12] = {1.0, 1.5, 1.2, 11.0,
									 0.9, 1.1, 0.8, 2.0,
											 1.0, 1.0, 0.6, -1.0};
	double *tf;

	HANDLE_ERROR(hipMalloc((void**)&tf, sizeof(double)*12));
	HANDLE_ERROR(hipMemcpy(tf, tf_org, sizeof(double)*12, hipMemcpyHostToDevice));

	double *ox, *oy, *oz;
	HANDLE_ERROR(hipMalloc((void**)&ox, sizeof(double)*N));
	HANDLE_ERROR(hipMalloc((void**)&oy, sizeof(double)*N));
	HANDLE_ERROR(hipMalloc((void**)&oz, sizeof(double)*N));

	double *x, *y, *z;

	for (int i = 0; i < count; i++)
		{
			HANDLE_ERROR(hipEventCreate(&start[i]));
			HANDLE_ERROR(hipEventCreate(&stop[i]));

			HANDLE_ERROR(hipSetDevice(i));

			HANDLE_ERROR(hipEventRecord(start[i], (hipStream_t)i));
			
			HANDLE_ERROR(hipMalloc((void**)&x, sizeof(double)*N));
			HANDLE_ERROR(hipMalloc((void**)&y, sizeof(double)*N));			
			HANDLE_ERROR(hipMalloc((void**)&z, sizeof(double)*N));

			HANDLE_ERROR(hipMemcpy(x, org_x, sizeof(double)*N, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(y, org_y, sizeof(double)*N, hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(z, org_z, sizeof(double)*N, hipMemcpyHostToDevice));

			kernel<<<BLOCK, THREAD>>>(x, y, z, tf, ox, oy, oz);

			HANDLE_ERROR(hipEventRecord(stop[i], (hipStream_t)i));
			HANDLE_ERROR(hipEventSynchronize(stop[i]));
			HANDLE_ERROR(hipEventElapsedTime(&time, start[i], stop[i]));

			std::cout << "device: " << i << std::endl;
			std::cout << "time: " << time << std::endl;

			HANDLE_ERROR(hipFree(x));
			HANDLE_ERROR(hipFree(y));
			HANDLE_ERROR(hipFree(z));

			HANDLE_ERROR(hipEventDestroy(start[i]));
			HANDLE_ERROR(hipEventDestroy(stop[i]));

		}

	HANDLE_ERROR(hipFree(tf));
	HANDLE_ERROR(hipFree(ox));
	HANDLE_ERROR(hipFree(oy));
	HANDLE_ERROR(hipFree(oz));

	return 0;

}