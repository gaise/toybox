#include <iostream>
#include <hip/hip_runtime.h>

int main()
{
	int *x = new int[10];

	for (int i = 0; i < 10; i++) {
		x[i] = i + 1;
	}

	int *d_x;

	hipMalloc((void**)&d_x, sizeof(int)*10);
	hipMemcpy(d_x, x, sizeof(int)*10, hipMemcpyHostToDevice);

	int *d_y = d_x + 5;

	int *y = new int[5];
	hipMemcpy(y, d_y, sizeof(int)*5, hipMemcpyDeviceToHost);

	std::cout << y[0] << " " << y[1] << " " << y[2] << " " << y[3] << " " << y[4] << std::endl;

	return 0;
}