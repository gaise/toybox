#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

void checkCudaError(hipError_t msg, int x)
{
  if (msg != hipSuccess) {
    fprintf(stderr, "line: %d %s\n", x, hipGetErrorString(msg));
    exit(1);
  }
  return;
}

int main()
{
  float *s, *dev_s;
  int i;
  std::chrono::time_point<std::chrono::system_clock> start, end;
  double time;
  
  s = (float *)malloc(sizeof(float)*1000);
  checkCudaError(hipMalloc((void**)&dev_s, sizeof(float)*1000), __LINE__);
  
  start = std::chrono::system_clock::now();

  checkCudaError(hipMemcpy(dev_s, s, sizeof(float)*1000, hipMemcpyHostToDevice), __LINE__);

  checkCudaError(hipDeviceSynchronize(), __LINE__);

  end = std::chrono::system_clock::now();

  time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

  free(s);
  checkCudaError(hipFree(dev_s), __LINE__);

  std::cout << time << "usec." << std::endl;

  return 0;
} 