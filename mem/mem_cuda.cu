#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define ITER 200

void checkCudaError(hipError_t msg, int x)
{
  if (msg != hipSuccess) {
    fprintf(stderr, "line: %d %s\n", x, hipGetErrorString(msg));
    exit(1);
  }
  return;
}

int main()
{
  float *s, *dev_s;
  int i, j;
  std::chrono::time_point<std::chrono::system_clock> start, end;
  double time;
  double array[ITER];
  
  for (i = 1; i <= ITER; i+=1) {
    s = (float *)malloc(sizeof(float)*i*100);
    checkCudaError(hipMalloc((void**)&dev_s, sizeof(float)*i*100), __LINE__);

    for (j = 0; j < i*100; j++) {
      s[j] = j;
    }
    
    start = std::chrono::system_clock::now();

    checkCudaError(hipMemcpy(dev_s, s, sizeof(float)*i*100, hipMemcpyHostToDevice), __LINE__);

    checkCudaError(hipDeviceSynchronize(), __LINE__);

    end = std::chrono::system_clock::now();

    time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

    array[i-1] = time;
	free(s);
	checkCudaError(hipFree(dev_s), __LINE__);
  }

  for (i = 0; i < ITER; i++) {
    std::cout << (i+1)*100 << " float : " << array[i] << "sec." << std::endl;
  }

  return 0;
} 
