#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define SIZE 1000

void checkCudaError(hipError_t msg, int x)
{
  if (msg != hipSuccess) {
    fprintf(stderr, "line: %d %s\n", x, hipGetErrorString(msg));
    exit(1);
  }
  return;
}

int main()
{
  float *s, *dev_s;
  int i;
  std::chrono::time_point<std::chrono::system_clock> start, end;
  double time;
  
  s = (float *)malloc(sizeof(float)*SIZE);
  for (i = 0; i < SIZE; i++) {
    s[i] = i;
  }
  checkCudaError(hipMalloc((void**)&dev_s, sizeof(float)*SIZE), __LINE__);
  
  start = std::chrono::system_clock::now();

  checkCudaError(hipMemcpy(dev_s, s, sizeof(float)*SIZE, hipMemcpyHostToDevice), __LINE__);

  checkCudaError(hipDeviceSynchronize(), __LINE__);

  end = std::chrono::system_clock::now();

  time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

  free(s);
  checkCudaError(hipFree(dev_s), __LINE__);

  std::cout << time << "usec." << std::endl;

  return 0;
} 